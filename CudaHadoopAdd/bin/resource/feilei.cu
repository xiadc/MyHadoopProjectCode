
#include <hip/hip_runtime.h>
extern "C"
__global__ void feilei(int n, float *hostInputA, float *hostInputB,float *result) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n) {
       for(int j = 0; j < n; j++){
       if(hostInputA[j]==1.70141E38f){ //如果chang_tile[j/4]的值是无效值，则无用值赋为0      
          result[j] = hostInputA[j];
          continue;
       }
      if(hostInputA[j]>=4500.0f){
       result[j] = 1.0f;  //山地类型1
       } else if(hostInputA[j]>=3500.0f && hostInputA[j]<4500.0f){
          result[j]= 2.0f;   //山地类型2
       }  else if(hostInputA[j]>=2500.0f && hostInputA[j]<3500.0f){
          result[j]= 3.0f;   //山地类型3
       } else if(hostInputA[j]>=1500.0f && hostInputA[j]<2500.0f && hostInputB[j] >= 2.0f){
          result[j]= 4.0f;   //山地类型4
       }  else if(hostInputA[j]>=1500.0f && hostInputA[j]<1000.0f && hostInputB[j] >= 5.0f){
          result[j]= 5.0f;   //山地类型5
       } else if(hostInputA[j]>=300.0f && hostInputA[j]<1000.0f){
          result[j]= 6.0f;   //山地类型6
       } else{
         result[j] = 0.0f; //非山地
       }
    }
  }
}
