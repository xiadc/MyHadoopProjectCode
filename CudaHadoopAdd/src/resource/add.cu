
#include <hip/hip_runtime.h>
extern "C"
__global__ void add(int n, float *hostInputA, float *hostInputB,float *result) {
     
     
	int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
	
	int i = blockId * (blockDim.x * blockDim.y * blockDim.z) + (threadIdx.z * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
	  //  int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n) {
       result[i] = hostInputA[i] + hostInputB[i];
  }
}
